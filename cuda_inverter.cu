#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#define T 32

using namespace cv;
using namespace std;

__global__ void Inversion(unsigned char* image, unsigned char* image_inv, int size) {

	int pixel = blockIdx.x*blockDim.x+threadIdx.x;

	unsigned char mcolor = '255';
	if ( pixel < size)
	{
		image_inv[pixel] = mcolor-image[pixel];
	}  
}

int cuda_run(string input_video)
{
	VideoCapture inp_vcap(input_video);
	if(!inp_vcap.isOpened()) return -1;

	Mat img;
	inp_vcap >> img;
	bool isColor = (img.type() == CV_8UC3);
	Size imgsize = img.size();
	int width = imgsize.width;
	int height = imgsize.height;
	Mat img_invert(height,width,CV_8UC3,Scalar(0,0,0));

	unsigned char* charImg = img.data;
	unsigned char* newImg = img_invert.data;

	int uCharSize = height*width*3*sizeof(unsigned char);

	unsigned char *devImg,*devInv;

	int vecSize = height*width*3;
	int blocks = (vecSize+T-1)/T;

	VideoWriter writer;
    int codec = VideoWriter::fourcc('a', 'v', 'c', '1');  // select desired codec (must be available at runtime)
    double fps = 25.0;                          // framerate of the created video stream
    string filename = "output_video.mp4";             // name of the output video file
    writer.open(filename, codec, fps, img.size(), isColor);
    // check if we succeeded
    if (!writer.isOpened()) {
        cerr << "Could not open the output video file for write\n";
        return -1;
    }

	// Mat img = imread(imgfile,IMREAD_COLOR);

	while(inp_vcap.isOpened()){
		if (img.empty()){
  	      break;
		}

		hipMalloc((void**) &devImg, uCharSize);
		hipMalloc((void**) &devInv, uCharSize);

		hipMemcpy(devImg,charImg,uCharSize,hipMemcpyHostToDevice);
		hipMemcpy(devInv,newImg,uCharSize,hipMemcpyHostToDevice);

		Inversion<<<blocks,T>>>  (devImg,devInv,vecSize);

		hipMemcpy(charImg,devImg,uCharSize,hipMemcpyDeviceToHost);
		hipMemcpy(newImg,devInv,uCharSize,hipMemcpyDeviceToHost);

		hipFree(devImg);
		hipFree(devInv);
		
		Mat output = Mat(height,width,CV_8UC3, newImg);

		writer.write(output);
		// imshow("Inverted Image",output);

		if (waitKey(5) >= 0) break;
		
		inp_vcap >> img;
	}
	destroyAllWindows();
	return 0;
}

